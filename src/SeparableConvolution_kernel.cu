#include "hip/hip_runtime.h"
#include <THC.h>
#include <THCGeneral.h>

#define VEC_0(ARRAY) ((ARRAY).x)
#define VEC_1(ARRAY) ((ARRAY).y)
#define VEC_2(ARRAY) ((ARRAY).z)
#define VEC_3(ARRAY) ((ARRAY).w)

#define IDX_1(ARRAY, X)          ((ARRAY)[((X) * (ARRAY##_stride.x))])
#define IDX_2(ARRAY, X, Y)       ((ARRAY)[((X) * (ARRAY##_stride.x)) + ((Y) * (ARRAY##_stride.y))])
#define IDX_3(ARRAY, X, Y, Z)    ((ARRAY)[((X) * (ARRAY##_stride.x)) + ((Y) * (ARRAY##_stride.y)) + ((Z) * (ARRAY##_stride.z))])
#define IDX_4(ARRAY, X, Y, Z, W) ((ARRAY)[((X) * (ARRAY##_stride.x)) + ((Y) * (ARRAY##_stride.y)) + ((Z) * (ARRAY##_stride.z)) + ((W) * (ARRAY##_stride.w))])

#ifdef __cplusplus
	extern "C" {
#endif

__global__ void kernel_SeparableConvolution_updateOutput(
	const int n,
	const float* input1, const long4 input1_size, const long4 input1_stride,
	const float* input2, const long4 input2_size, const long4 input2_stride,
	const float* input3, const long4 input3_size, const long4 input3_stride,
	float* output, const long4 output_size, const long4 output_stride
) {
	int intIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (intIndex >= n) {
		return;
	}

	float dblOutput = 0.0;

	int intBatch = ( intIndex / VEC_3(output_size) / VEC_2(output_size) / VEC_1(output_size) ) % VEC_0(output_size);
	int intDepth = ( intIndex / VEC_3(output_size) / VEC_2(output_size)                      ) % VEC_1(output_size);
	int intY     = ( intIndex / VEC_3(output_size)                                           ) % VEC_2(output_size);
	int intX     = ( intIndex                                                                ) % VEC_3(output_size);

	for (int intFilterY = 0; intFilterY < 51; intFilterY += 1) {
		for (int intFilterX = 0; intFilterX < 51; intFilterX += 1) {
			dblOutput += IDX_4(input1, intBatch, intDepth, intY + intFilterY, intX + intFilterX) * IDX_4(input2, intBatch, intFilterY, intY, intX) * IDX_4(input3, intBatch, intFilterX, intY, intX);
		}
	}

	output[intIndex] = dblOutput;
}

void SeparableConvolution_kernel_forward(
	THCState* state,
	THCudaTensor* input1,
	THCudaTensor* input2,
	THCudaTensor* input3,
	THCudaTensor* output
) {
	int n = 0;

	n = THCudaTensor_nElement(state, output);
	kernel_SeparableConvolution_updateOutput<<< (n + 512 - 1) / 512, 512, 0, THCState_getCurrentStream(state) >>>(
		n,
		THCudaTensor_data(state, input1), make_long4(input1->size[0], input1->size[1], input1->size[2], input1->size[3]), make_long4(input1->stride[0], input1->stride[1], input1->stride[2], input1->stride[3]),
		THCudaTensor_data(state, input2), make_long4(input2->size[0], input2->size[1], input2->size[2], input2->size[3]), make_long4(input2->stride[0], input2->stride[1], input2->stride[2], input2->stride[3]),
		THCudaTensor_data(state, input3), make_long4(input3->size[0], input3->size[1], input3->size[2], input3->size[3]), make_long4(input3->stride[0], input3->stride[1], input3->stride[2], input3->stride[3]),
		THCudaTensor_data(state, output), make_long4(output->size[0], output->size[1], output->size[2], output->size[3]), make_long4(output->stride[0], output->stride[1], output->stride[2], output->stride[3])
	);

	THCudaCheck(hipGetLastError());
}

#ifdef __cplusplus
	}
#endif